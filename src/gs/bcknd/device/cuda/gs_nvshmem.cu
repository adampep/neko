/*
 Copyright (c) 2024, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <device/device_config.h>
#include <device/cuda/check.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <comm/comm.h>
#include "gs_kernels.h"
#include "gs_nvshmem_kernels.h"

extern "C" {

  void cudamalloc_nvshmem(void** ptr, size_t size)
  {
    *ptr = nvshmem_malloc(size);
    CUDA_CHECK(hipGetLastError());
    hipMemset(*ptr, 0, size);
    CUDA_CHECK(hipGetLastError());
  }    
  
  void cudafree_nvshmem(void** ptr, size_t size)
  {
    nvshmem_free(*ptr);
    CUDA_CHECK(hipGetLastError());
  }
  
  void cuda_gs_pack_and_push(void *u_d, void *sbuf_d, void *sdof_d,
                             int soffset, int n, hipStream_t stream,
                             int srank,  void *rbuf_d, int roffset, int* remote_offset,
			     int rrank, uint64_t counter, void* notifyDone, void* notifyReady,
			     int iter)
  {
    
    if(remote_offset[iter-1] == -1)
    {
      MPI_Sendrecv(&roffset, 1, MPI_INT,
                   rrank, 0,
                   &(remote_offset[iter-1]), 1, MPI_INT,
                   srank, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
    
    const int nthrds = 1024;
    const int nblcks = (n+nthrds-1)/nthrds;
      
    // TO DO investigate merging following 2 kernels (and also unpack).  
    gs_pack_kernel<real>
      <<<nblcks, nthrds, 0, stream>>>((real *) u_d, (real *) sbuf_d + soffset,
                                      (int *) sdof_d + soffset, n);
    
    pushShmemKernel<real>
      <<<nblcks,nthrds,0,stream>>>((real *) rbuf_d + remote_offset[iter-1],
                                   (real *) sbuf_d + soffset,
                                   (int *) sdof_d + soffset,
                                   srank, rrank, n, counter,
                                   (uint64_t*) notifyDone,
                                   (uint64_t*) notifyReady);
    CUDA_CHECK(hipGetLastError());
  }

  void cuda_gs_pack_and_push_wait(hipStream_t stream, int counter, void* notifyDone)
  {
    pushShmemKernelWait<<<1,1,0,stream>>>(counter,(uint64_t*) notifyDone);
    CUDA_CHECK(hipGetLastError());
  }
}