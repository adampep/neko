#include "hip/hip_runtime.h"
/*
 Copyright (c) 2024, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <device/device_config.h>
#include <device/cuda/check.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <comm/comm.h>
#include "gs_kernels.h"

extern "C" {

  void cudamalloc_nvshmem(void** ptr, size_t size)
  {
    nvshmemx_init_attr_t attr;
    attr.mpi_comm = &NEKO_COMM;
    if (nvshmemx_init_status()==NVSHMEM_STATUS_NOT_INITIALIZED)
    {
        nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
        //        int provided;
        //        nvshmem_init_thread(NVSHMEM_THREAD_MULTIPLE, &provided);
    }
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    *ptr = nvshmem_malloc(size);
    CUDA_CHECK(hipGetLastError());
    hipMemset(*ptr,0,size);
    CUDA_CHECK(hipGetLastError());
  }    
  
  void cudafree_nvshmem(void** ptr, size_t size)
  {
    nvshmem_free(*ptr);
      CUDA_CHECK(hipGetLastError());
  }
  
  __global__ void pushShmemKernel(real* dest, real* src, int* dof,
                                  int destRank, int srcRank, int n,
                                  uint64_t counter, uint64_t* notifyDone,
                                  uint64_t* notifyReady)
  {
    //TO DO: 1 block transfers seem best from initial investigations, check this more thoroughly
    size_t numBlocksForTransfer = 1; 
    if(blockIdx.x < numBlocksForTransfer)
    {
      size_t n_per_block = n/numBlocksForTransfer;
      size_t block_offset = n_per_block*blockIdx.x;
      size_t dataSize = blockIdx.x != (numBlocksForTransfer - 1) ? n_per_block : max(n - block_offset, n_per_block);
      // Notify ready to sending rank, and wait until recieving rank is ready
      if (threadIdx.x == 0) {
        nvshmemx_signal_op(notifyReady, counter, NVSHMEM_SIGNAL_SET, srcRank);
        nvshmem_signal_wait_until(notifyReady, NVSHMEM_CMP_EQ, counter);
      }
      __syncthreads();
          
      // Push data
	  nvshmemx_double_put_signal_nbi_block(dest + block_offset, src +
                                               block_offset, dataSize,
                                               notifyDone, counter,
                                               NVSHMEM_SIGNAL_SET, destRank);
    }
  }

  __global__ void pushShmemKernelWait(uint64_t counter, uint64_t* notifyDone)
  {
    // Notify done to receiving rank, and wait for data from sending rank
    if (blockIdx.x==0 && threadIdx.x == 0) {
      nvshmem_signal_wait_until(notifyDone, NVSHMEM_CMP_EQ, counter);
    }
  }
    
  void cuda_gs_pack_and_push(void *u_d, void *sbuf_d, void *sdof_d,
                             int soffset, int n, hipStream_t stream,
                             int srank,  void *rbuf_d, int roffset, int* remote_offset,
			     int rrank, int counter, void* notifyDone, void* notifyReady,
			     int iter)
  {
    
    if(remote_offset[iter-1] == -1)
    {
      MPI_Sendrecv(&roffset, 1, MPI_INT,
                   rrank, 0,
                   &(remote_offset[iter-1]), 1, MPI_INT,
                   srank, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
    
    const int nthrds = 1024;
    const int nblcks = (n+nthrds-1)/nthrds;
      
    // TO DO investigate merging following 2 kernels (and also unpack).  
    gs_pack_kernel<real>
      <<<nblcks, nthrds, 0, stream>>>((real *) u_d, (real *) sbuf_d + soffset,
                                      (int *) sdof_d + soffset, n);
    
    pushShmemKernel<<<nblcks,nthrds,0,stream>>>((real *) rbuf_d + remote_offset[iter-1],
                                                (real *) sbuf_d + soffset,
                                                (int *) sdof_d + soffset,
                                                srank, rrank, n, counter,
                                                (uint64_t*) notifyDone,
                                                (uint64_t*) notifyReady);
    CUDA_CHECK(hipGetLastError());
  }

  void cuda_gs_pack_and_push_wait(hipStream_t stream, int counter, void* notifyDone)
  {
    pushShmemKernelWait<<<1,1,0,stream>>>(counter,(uint64_t*) notifyDone);
    CUDA_CHECK(hipGetLastError());
  }
}